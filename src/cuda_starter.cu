#include <hip/hip_runtime.h>



__global__ void add_vectors_kernel(double *a, double *b, double *c, size_t N) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < N) c[id] = a[id] + b[id];
}

void add_vectors(double *d_a, double *d_b, double *d_c, size_t N) {
    dim3 blockSize(256, 1, 1);
    dim3 gridSize((N - 1) / blockSize.x + 1, 1, 1);

    add_vectors_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, N);
}