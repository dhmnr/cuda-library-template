#include <hip/hip_runtime.h>


#include <stdio.h>

int cudaErrorCheck(hipError_t err, const char *errString, bool isFatal = true) {
    if (err != hipSuccess) {
        fprintf(stderr, "%s \n\tError %s : %s\n", errString, hipGetErrorName(err),
                hipGetErrorString(err));
        if (isFatal)
            exit(EXIT_FAILURE);
        else
            return 1;
    }
    return 0;
}