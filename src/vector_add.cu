#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#include "cuda_starter/utils.h"

__global__ void add_vectors_kernel(double *a, double *b, double *c, size_t N) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < N) c[id] = a[id] + b[id];
}

void add_vectors(double *d_a, double *d_b, double *d_c, size_t N) {
    dim3 blockSize(256, 1, 1);
    dim3 gridSize((N - 1) / blockSize.x + 1, 1, 1);

    add_vectors_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, N);

    hipError_t err = hipDeviceSynchronize();
    cudaErrorCheck(err, "Failed to execute add_vectors_kernel");
}

void add_vectors_with_copy(double *h_A, double *h_B, double *h_C, size_t N) {
    hipError_t err;
    size_t size = N * sizeof(double);

    double *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    cudaErrorCheck(err, "Failed to allocate device vector A");

    double *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    cudaErrorCheck(err, "Failed to allocate device vector B");

    double *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    cudaErrorCheck(err, "Failed to allocate device vector C");

    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    cudaErrorCheck(err, "Failed to copy vector A from host to device");

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    cudaErrorCheck(err, "Failed to copy vector B from host to device");

    add_vectors(d_A, d_B, d_C, N);

    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    cudaErrorCheck(err, "Failed to copy vector C from device to host");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
